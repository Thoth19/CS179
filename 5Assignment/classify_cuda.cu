#include "hip/hip_runtime.h"
#include <cassert>
#include <cmath>
#include <cstdio>
#include <hip/hip_runtime.h>
#include "classify_cuda.cuh"

/*
 * Arguments:
 * data: Memory that contains both the review LSA coefficients and the labels.
 *       Format is the last element is -1 if not a restaurant and 1 if it is.
            elements 0-49 are the bag of words values
 * batch_size: Size of mini-batch, how many elements to process at once
 * step_size: Step size for gradient descent. Tune this as needed. 1.0 is sane
 *            default.
 * weights: Pointer to weights vector of length REVIEW_DIM (50).
 * error: Pointer to a single int used to describe the error for the batch.
 *         An output variable for the kernel. It is the misclassification rate
 *         in the batch to errors.
 */
__global__
void trainLogRegKernel(
    float *data,
    int batch_size,
    float step_size,
	float *weights,
    int *error)
{
    unsigned int thread_index = blockIdx.x * blockDim.x + threadIdx.x;
    float dot_prod;
    float y_dot_prod[REVIEW_DIM];
    float grad;
    float grad_coeff;
    int i;
    int misclass[batch_size];
    float weight_delta[batch_size][REVIEW_DIM];
    while(thread_index < batch_size)
    {
        // compute the dot_prod
        i = 0;
        while(i < REVIEW_DIM)
        {
            dot_prod += weights[i] * data[thread_index][i];
            i += 1;
        }

        //compute the number of misclassifications by comparing the signs
        // of y_n and the weight * data
        y_dot_prod = data[thread_index][REVIEW_DIM] * dot_prod;
        if (y_dot_prod < 0)
        {
            misclass[thread_index] += 1;
        }

        // compute gradient/updated weight values
        i = 0;
        grad_coeff = (-1/ batch_size) / (1 + exp(y_dot_prod));
        while (i < REVIEW_DIM)
        {
            grad[i] = grad_coeff * (data[thread_index][REVIEW_DIM] * data[thread_index][i]);
            weight_delta[thread_index][i] -= step_size * grad[i];
        }
        thread_index += (blockDim.x * gridDim.x);
    }
    // Do the atomic adds
    thread_index = blockIdx.x * blockDim.x + threadIdx.x;
    while(thread_index < batch_size)
    {
        for (i = 0; i < REVIEW_DIM; ++i)
        {
            atomicAdd(&weights[i], - weight_delta[thread_index][i]);
        }
        atomicAdd(error, misclass[thread_index]);
        thread_index += (blockDim.x * gridDim.x);
    }
}

/*
 * All parameters have the same meaning as in docstring for trainLogRegKernel.
 * Notably, cudaClassify returns a float that quantifies the error in the
 * minibatch. This error should go down as more training occurs.
 */
float cudaClassify(
    float *data,
    int batch_size, 
    float step_size,
    float *weights, 
    hipStream_t stream)
{
    int block_size = (batch_size < 1024) ? batch_size : 1024;

    // grid_size = CEIL(batch_size / block_size)
    int grid_size = (batch_size + block_size - 1) / block_size;
    int shmem_bytes = 0;

    int *d_errors;
    hipMalloc(&d_errors, sizeof(int));
    hipMemset(d_errors, 0, sizeof(int));

    trainLogRegKernel<<<grid_size, block_size, shmem_bytes, stream>>>(
        data,
        batch_size,
        step_size,
        weights,
        d_errors);

    int h_errors = 0;
    hipMemcpy(&h_errors, d_errors, sizeof(int), hipMemcpyDefault);
    hipFree(d_errors);
    return h_errors;
}
